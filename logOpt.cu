#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <IL/il.h>
#include <hip/hip_runtime.h>

__global__ void cudaLaplacienDeGaussienne(unsigned char *donneesSrc, unsigned char *donneesDst, int largeur, int hauteur, int bpp, int blockSizeX, int blockSizeY) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    
    __shared__ unsigned char sharedBlock[blockSizeY][blockSizeX][4];

    int masque[5][5] = {
        {0, 0, -1, 0, 0},
        {0, -1, -2, -1, 0},
        {-1, -2, 16, -2, -1},
        {0, -1, -2, -1, 0},
        {0, 0, -1, 0, 0}
    };

    if (x < blockSizeX || y < blockSizeY || x + blockSizeX >= largeur || y + blockSizeY >= hauteur) {
        for (int c = 0; c < bpp; ++c) {
            donneesDst[(y * largeur + x) * bpp + c] = donneesSrc[(y * largeur + x) * bpp + c];
        }
        return;
    }

    for (int c = 0; c < bpp; ++c) {
        int somme = 0;

        if (threadIdx.x < blockSizeX && threadIdx.y < blockSizeY) {
            sharedBlock[threadIdx.y][threadIdx.x][c] = donneesSrc[((y - blockSizeY + threadIdx.y) * largeur + (x - blockSizeX + threadIdx.x)) * bpp + c];
            __syncthreads();
        }

        if (threadIdx.x >= blockSizeX/2 && threadIdx.y >= blockSizeY/2 && threadIdx.x < blockSizeX/2 + blockSizeX && threadIdx.y < blockSizeY/2 + blockSizeY) {
            for (int fy = -2; fy <= 2; ++fy) {
                for (int fx = -2; fx <= 2; ++fx) {
                    somme += masque[fy + 2][fx + 2] * sharedBlock[threadIdx.y - blockSizeY/2 + fy][threadIdx.x - blockSizeX/2 + fx][c];
                }
            }
        }

        donneesDst[(y * largeur + x) * bpp + c] = fminf(fmaxf(somme, 0), 255);
    }
}

void LaplacienDeGaussienneGPU(unsigned char *donnees, unsigned char *nouvellesDonnees, int largeur, int hauteur, int bpp, int iterations, int blockSizeX, int blockSizeY) {
    unsigned char *donneesSrc = donnees;

    unsigned char *donneesSrcDevice;
    unsigned char *donneesDstDevice;

    size_t size = largeur * hauteur * bpp * sizeof(unsigned char);

    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void **)&donneesSrcDevice, size);
    if (cudaStatus != hipSuccess) {
        std::cerr << "Erreur lors de l'allocation de mémoire sur le GPU pour les données source" << std::endl;
        return;
    }

    cudaStatus = hipMalloc((void **)&donneesDstDevice, size);
    if (cudaStatus != hipSuccess) {
        std::cerr << "Erreur lors de l'allocation de mémoire sur le GPU pour les données de destination" << std::endl;
        hipFree(donneesSrcDevice);
        return;
    }

    cudaStatus = hipMemcpy(donneesSrcDevice, donneesSrc, size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        std::cerr << "Erreur lors de la copie des données source de l'hôte vers le GPU" << std::endl;
        hipFree(donneesSrcDevice);
        hipFree(donneesDstDevice);
        return;
    }

    dim3 blockSize(blockSizeX, blockSizeY);
    dim3 gridSize((largeur + blockSize.x - 1) / blockSize.x, (hauteur + blockSize.y - 1) / blockSize.y);

    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    for (int i = 0; i < iterations; ++i) {
        if (i % 2 == 0) {
            cudaLaplacienDeGaussienne<<<gridSize, blockSize, 0, stream1>>>(donneesSrcDevice, donneesDstDevice, largeur, hauteur, bpp, blockSizeX, blockSizeY);
        } else {
            cudaLaplacienDeGaussienne<<<gridSize, blockSize, 0, stream2>>>(donneesDstDevice, donneesSrcDevice, largeur, hauteur, bpp, blockSizeX, blockSizeY);
        }

        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            std::cerr << "Erreur lors de l'exécution du kernel CUDA" << std::endl;
            hipFree(donneesSrcDevice);
            hipFree(donneesDstDevice);
            hipStreamDestroy(stream1);
            hipStreamDestroy(stream2);
            return;
        }
    }

    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    if (iterations % 2 == 1) {
        hipMemcpyAsync(nouvellesDonnees, donneesSrcDevice, size, hipMemcpyDeviceToHost);
    } else {
        hipMemcpyAsync(nouvellesDonnees, donneesDstDevice, size, hipMemcpyDeviceToHost);
    }

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "Erreur lors de la copie des données de destination du GPU vers l'hôte" << std::endl;
        hipFree(donneesSrcDevice);
        hipFree(donneesDstDevice);
        hipStreamDestroy(stream1);
        hipStreamDestroy(stream2);
        return;
    }

    hipFree(donneesSrcDevice);
    hipFree(donneesDstDevice);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
}

int main(int argc, char *argv[]) {
    unsigned int image;

    // Initialisation de DevIL
    ilInit();
    ilGenImages(1, &image);
    ilBindImage(image);
    ilLoadImage(argv[1]);

    int largeur, hauteur, bpp, format;

    largeur = ilGetInteger(IL_IMAGE_WIDTH);
    hauteur = ilGetInteger(IL_IMAGE_HEIGHT);
    bpp = ilGetInteger(IL_IMAGE_BYTES_PER_PIXEL);
    format = ilGetInteger(IL_IMAGE_FORMAT);

    unsigned char *donnees = ilGetData();
    unsigned char *nouvellesDonnees = new unsigned char[largeur * hauteur * bpp];

    int iterations = std::stoi(argv[2]);
    int blockSizeX = std::stoi(argv[3]);
    int blockSizeY = std::stoi(argv[4]);

    // Appliquer le filtre LoG sur l'image avec le nombre d'itérations spécifié
    LaplacienDeGaussienneGPU(donnees, nouvellesDonnees, largeur, hauteur, bpp, iterations, blockSizeX, blockSizeY);

    // Mettre à jour l'image avec les données traitées
    ilTexImage(largeur, hauteur, 1, bpp, format, IL_UNSIGNED_BYTE, nouvellesDonnees);

    // Activer l'écrasement de fichier lors de la sauvegarde
    ilEnable(IL_FILE_OVERWRITE);
    ilSaveImage(argv[5]);

    // Supprimer l'image de la mémoire de DevIL
    ilDeleteImages(1, &image);

    // Libérer la mémoire allouée pour les nouvelles données de l'image
    delete[] nouvellesDonnees;

    return 0;
}
