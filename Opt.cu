#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <IL/il.h>
#include <hip/hip_runtime.h>

void normaliserNoyauGaussien(float noyauGaussien[5][5]) {
    float somme = 0;
    for (int i = 0; i < 5; ++i) {
        for (int j = 0; j < 5; ++j) {
            somme += noyauGaussien[i][j];
        }
    }
    for (int i = 0; i < 5; ++i) {
        for (int j = 0; j < 5; ++j) {
            noyauGaussien[i][j] /= somme;
        }
    }
}

__global__ void applyGaussianBlur(unsigned char *donneesSrc, unsigned char *donneesDst, int largeur, int hauteur, int bpp, float *noyauGaussien) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < 2 || y < 2 || x + 2 >= largeur || y + 2 >= hauteur) {
        for (int c = 0; c < bpp; ++c) {
            donneesDst[(y * largeur + x) * bpp + c] = donneesSrc[(y * largeur + x) * bpp + c];
        }
        return;
    }

    extern __shared__ float sharedNoyauGaussien[];
    if (threadIdx.x < 5 && threadIdx.y < 5) {
        sharedNoyauGaussien[threadIdx.y * 5 + threadIdx.x] = noyauGaussien[threadIdx.y * 5 + threadIdx.x];
    }
    __syncthreads();

    for (int c = 0; c < bpp; ++c) {
        float somme = 0;
        for (int ky = -2; ky <= 2; ++ky) {
            for (int kx = -2; kx <= 2; ++kx) {
                somme += donneesSrc[((y + ky) * largeur + (x + kx)) * bpp + c] * sharedNoyauGaussien[(ky + 2) * 5 + (kx + 2)];
            }
        }
        donneesDst[(y * largeur + x) * bpp + c] = somme;
    }
}

void flouGaussienGPU(unsigned char *donnees, unsigned char *nouvellesDonnees, int largeur, int hauteur, int bpp, int iterations) {
    float noyauGaussien[5][5] = {
        {1.0f, 4.0f,  7.0f,  4.0f, 1.0f},
        {4.0f, 16.0f, 26.0f, 16.0f, 4.0f},
        {7.0f, 26.0f, 41.0f, 26.0f, 7.0f},
        {4.0f, 16.0f, 26.0f, 16.0f, 4.0f},
        {1.0f, 4.0f,  7.0f,  4.0f, 1.0f}
    };

    normaliserNoyauGaussien(noyauGaussien);

    float *devNoyauGaussien;
    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)&devNoyauGaussien, 5 * 5 * sizeof(float));
    if (cudaStatus != hipSuccess) {
        std::cerr << "Erreur lors de l'allocation de mémoire sur le GPU pour le noyau gaussien" << std::endl;
        return;
    }

    cudaStatus = hipMemcpy(devNoyauGaussien, noyauGaussien, 5 * 5 * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        std::cerr << "Erreur lors de la copie du noyau gaussien de l'hôte vers le GPU" << std::endl;
        hipFree(devNoyauGaussien);
        return;
    }

    dim3 blockSize(16, 16);
    dim3 gridSize((largeur + blockSize.x - 1) / blockSize.x, (hauteur + blockSize.y - 1) / blockSize.y);

    unsigned char *devDonneesSrc, *devDonneesDst;
    
    cudaStatus = hipMalloc((void**)&devDonneesSrc, largeur * hauteur * bpp * sizeof(unsigned char));
    if (cudaStatus != hipSuccess) {
        std::cerr << "Erreur lors de l'allocation de mémoire sur le GPU pour les données source" << std::endl;
        hipFree(devNoyauGaussien);
        return;
    }

    cudaStatus = hipMalloc((void**)&devDonneesDst, largeur * hauteur * bpp * sizeof(unsigned char));
    if (cudaStatus != hipSuccess) {
        std::cerr << "Erreur lors de l'allocation de mémoire sur le GPU pour les données de destination" << std::endl;
        hipFree(devDonneesSrc);
        hipFree(devNoyauGaussien);
        return;
    }

    cudaStatus = hipMemcpy(devDonneesSrc, donnees, largeur * hauteur * bpp * sizeof(unsigned char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        std::cerr << "Erreur lors de la copie des données source de l'hôte vers le GPU" << std::endl;
        hipFree(devDonneesSrc);
        hipFree(devDonneesDst);
        hipFree(devNoyauGaussien);
        return;
    }

    hipStream_t stream;
    hipStreamCreate(&stream);

    for (int it = 0; it < iterations; ++it) {
        if (it < iterations - 1) {
            applyGaussianBlur<<<gridSize, blockSize, 5 * 5 * sizeof(float), stream>>>(devDonneesSrc, devDonneesDst, largeur, hauteur, bpp, devNoyauGaussien);
            cudaStatus = hipGetLastError();
            if (cudaStatus != hipSuccess) {
                std::cerr << "Erreur lors de l'exécution du kernel CUDA : " << hipGetErrorString(cudaStatus) << std::endl;
                hipFree(devDonneesSrc);
                hipFree(devDonneesDst);
                hipFree(devNoyauGaussien);
                return;
            }
            unsigned char *temp = devDonneesSrc;
            devDonneesSrc = devDonneesDst;
            devDonneesDst = temp;
        }
        else {
            applyGaussianBlur<<<gridSize, blockSize, 5 * 5 * sizeof(float), stream>>>(devDonneesSrc, devDonneesDst, largeur, hauteur, bpp, devNoyauGaussien);
            cudaStatus = hipGetLastError();
            if (cudaStatus != hipSuccess) {
                std::cerr << "Erreur lors de l'exécution du kernel CUDA : " << hipGetErrorString(cudaStatus) << std::endl;
                hipFree(devDonneesSrc);
                hipFree(devDonneesDst);
                hipFree(devNoyauGaussien);
                return;
            }
        }
    }

    hipMemcpyAsync(nouvellesDonnees, devDonneesDst, largeur * hauteur * bpp * sizeof(unsigned char), hipMemcpyDeviceToHost, stream);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "Erreur lors de la copie des données de destination du GPU vers l'hôte : " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(devDonneesSrc);
        hipFree(devDonneesDst);
        hipFree(devNoyauGaussien);
        return;
    }

    hipStreamDestroy(stream);

    hipFree(devDonneesSrc);
    hipFree(devDonneesDst);
    hipFree(devNoyauGaussien);
}

int main(int argc, char *argv[]) {
    unsigned int image;

    ilInit();
    ilGenImages(1, &image);
    ilBindImage(image);
    ilLoadImage(argv[1]);

    int largeur, hauteur, bpp, format;

    largeur = ilGetInteger(IL_IMAGE_WIDTH);
    hauteur = ilGetInteger(IL_IMAGE_HEIGHT);
    bpp = ilGetInteger(IL_IMAGE_BYTES_PER_PIXEL);
    format = ilGetInteger(IL_IMAGE_FORMAT);

    unsigned char *donnees = ilGetData();
    unsigned char *nouvellesDonnees = new unsigned char[largeur * hauteur * bpp];

    int iterations = std::stoi(argv[2]);

    flouGaussienGPU(donnees, nouvellesDonnees, largeur, hauteur, bpp, iterations);

    ilTexImage(largeur, hauteur, 1, bpp, format, IL_UNSIGNED_BYTE, nouvellesDonnees);

    ilEnable(IL_FILE_OVERWRITE);
    ilSaveImage(argv[3]);

    ilDeleteImages(1, &image);
    delete[] nouvellesDonnees;

    return 0;
}

