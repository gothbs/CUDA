#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <IL/il.h>
#include <hip/hip_runtime.h>

__global__ void SobelCUDA(const unsigned char *input, unsigned char *output, int width, int height, int bpp) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int sobelX[3][3] = { {-1, 0, 1}, {-2, 0, 2}, {-1, 0, 1} };
        int sobelY[3][3] = { {-1, -2, -1}, {0, 0, 0}, {1, 2, 1} };

        int index = (y * width + x) * bpp;

        if (x >= 1 && y >= 1 && x + 1 < width && y + 1 < height) {
            for (int c = 0; c < bpp; ++c) {
                int gradX = 0;
                int gradY = 0;
                for (int ky = -1; ky <= 1; ++ky) {
                    for (int kx = -1; kx <= 1; ++kx) {
                        int pixel = input[((y + ky) * width + (x + kx)) * bpp + c];
                        gradX += pixel * sobelX[ky + 1][kx + 1];
                        gradY += pixel * sobelY[ky + 1][kx + 1];
                    }
                }
                int magnitude = sqrt(gradX * gradX + gradY * gradY);
                output[index + c] = magnitude > 255 ? 255 : magnitude;
            }
        }
        else {
            for (int c = 0; c < bpp; ++c) {
                output[index + c] = input[index + c];
            }
        }
    }
}

void SobelGPU(unsigned char *donnees, unsigned char *nouvellesDonnees, int largeur, int hauteur, int bpp, int iterations, int blockSizeX, int blockSizeY) {
    unsigned char *donneesSrc = donnees;
    unsigned char *donneesDst = nouvellesDonnees;

    unsigned char *donneesSrcDevice;
    unsigned char *donneesDstDevice;

    size_t size = largeur * hauteur * bpp * sizeof(unsigned char);

    hipMalloc((void**)&donneesSrcDevice, size);
    hipMalloc((void**)&donneesDstDevice, size);

    hipMemcpy(donneesSrcDevice, donneesSrc, size, hipMemcpyHostToDevice);

    dim3 blockDim(blockSizeX, blockSizeY);
    dim3 gridDim((largeur + blockDim.x - 1) / blockDim.x, (hauteur + blockDim.y - 1) / blockDim.y);

    for (int i = 0; i < iterations; ++i) {
        SobelCUDA<<<gridDim, blockDim>>>(donneesSrcDevice, donneesDstDevice, largeur, hauteur, bpp);

        hipError_t cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            std::cerr << "Erreur lors de l'exécution du kernel CUDA : " << hipGetErrorString(cudaStatus) << std::endl;
            hipFree(donneesSrcDevice);
            hipFree(donneesDstDevice);
            return;
        }

        hipMemcpy(donneesSrcDevice, donneesDstDevice, size, hipMemcpyDeviceToDevice);

    }

    hipMemcpy(nouvellesDonnees, donneesDstDevice, size, hipMemcpyDeviceToHost);

    hipFree(donneesSrcDevice);
    hipFree(donneesDstDevice);
}

int main(int argc, char *argv[]) {
    unsigned int image;

    ilInit();
    ilGenImages(1, &image);
    ilBindImage(image);
    ilLoadImage(argv[1]);

    int largeur, hauteur, bpp, format;

    largeur = ilGetInteger(IL_IMAGE_WIDTH);
    hauteur = ilGetInteger(IL_IMAGE_HEIGHT);
    bpp = ilGetInteger(IL_IMAGE_BYTES_PER_PIXEL);
    format = ilGetInteger(IL_IMAGE_FORMAT);

    unsigned char *donnees = ilGetData();
    unsigned char *nouvellesDonnees = new unsigned char[largeur * hauteur * bpp];

    int iterations = std::stoi(argv[2]);
    int blockSizeX = std::stoi(argv[3]);
    int blockSizeY = std::stoi(argv[4]);

    // Appliquer l'opérateur de Sobel sur l'image avec le nombre d'itérations spécifié
    SobelGPU(donnees, nouvellesDonnees, largeur, hauteur, bpp, iterations, blockSizeX, blockSizeY);

    // Mettre à jour l'image avec les données traitées
    ilTexImage(largeur, hauteur, 1, bpp, format, IL_UNSIGNED_BYTE, nouvellesDonnees);

    // Activer l'écrasement de fichier lors de la sauvegarde
    ilEnable(IL_FILE_OVERWRITE);
    ilSaveImage(argv[5]);

    // Supprimer l'image de la mémoire de DevIL
    ilDeleteImages(1, &image);

    // Libérer la mémoire allouée pour les nouvelles données de l'image
    delete[] nouvellesDonnees;

    return 0;
}
