#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <IL/il.h>
#include <hip/hip_runtime.h>

void normaliserNoyauGaussien(float noyauGaussien[5][5] ) {
    float somme = 0;
    for (int i = 0; i < 5; ++i) {
        for (int j = 0; j < 5; ++j) {
            somme += noyauGaussien[i][j];
        }
    }
    for (int i = 0; i < 5; ++i) {
        for (int j = 0; j < 5; ++j) {
            noyauGaussien[i][j] /= somme;
        }
    }
}

__global__ void applyGaussianBlur(unsigned char *donneesSrc, unsigned char *donneesDst, int largeur, int hauteur, int bpp, float *noyauGaussien) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < 2 || y < 2 || x + 2 >= largeur || y + 2 >= hauteur) {
        for (int c = 0; c < bpp; ++c) {
            donneesDst[(y * largeur + x) * bpp + c] = donneesSrc[(y * largeur + x) * bpp + c];
        }
        return;
    }

    for (int c = 0; c < bpp; ++c) {
        float somme = 0;
        for (int ky = -2; ky <= 2; ++ky) {
            for (int kx = -2; kx <= 2; ++kx) {
                somme += donneesSrc[((y + ky) * largeur + (x + kx)) * bpp + c] * noyauGaussien[(ky + 2) * 5 + (kx + 2)];
            }
        }
        donneesDst[(y * largeur + x) * bpp + c] = somme;
    }
}

void flouGaussienGPU(unsigned char *donnees, unsigned char *nouvellesDonnees, int largeur, int hauteur, int bpp, int iterations) {
    float noyauGaussien[5][5] = {
        {1.0f, 4.0f,  7.0f,  4.0f, 1.0f},
        {4.0f, 16.0f, 26.0f, 16.0f, 4.0f},
        {7.0f, 26.0f, 41.0f, 26.0f, 7.0f},
        {4.0f, 16.0f, 26.0f, 16.0f, 4.0f},
        {1.0f, 4.0f,  7.0f,  4.0f, 1.0f}
    };

    normaliserNoyauGaussien(noyauGaussien);

    float *devNoyauGaussien;
    hipMalloc((void**)&devNoyauGaussien, 5 * 5 * sizeof(float));
    hipMemcpy(devNoyauGaussien, noyauGaussien, 5 * 5 * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((largeur + blockSize.x - 1) / blockSize.x, (hauteur + blockSize.y - 1) / blockSize.y);

    unsigned char *devDonneesSrc, *devDonneesDst;
    hipMalloc((void**)&devDonneesSrc, largeur * hauteur * bpp * sizeof(unsigned char));
    hipMalloc((void**)&devDonneesDst, largeur * hauteur * bpp * sizeof(unsigned char));

    hipMemcpy(devDonneesSrc, donnees, largeur * hauteur * bpp * sizeof(unsigned char), hipMemcpyHostToDevice);

    for (int it = 0; it < iterations; ++it) {
        if (it < iterations - 1) {
            applyGaussianBlur<<<gridSize, blockSize>>>(devDonneesSrc, devDonneesDst, largeur, hauteur, bpp, devNoyauGaussien);
            unsigned char *temp = devDonneesSrc;
            devDonneesSrc = devDonneesDst;
            devDonneesDst = temp;
        }
        else {
            applyGaussianBlur<<<gridSize, blockSize>>>(devDonneesSrc, devDonneesDst, largeur, hauteur, bpp, devNoyauGaussien);
        }
    }

    hipMemcpy(nouvellesDonnees, devDonneesDst, largeur * hauteur * bpp * sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipFree(devDonneesSrc);
    hipFree(devDonneesDst);
    hipFree(devNoyauGaussien);
}

int main(int argc, char *argv[]) {
    unsigned int image;

    ilInit();
    ilGenImages(1, &image);
    ilBindImage(image);
    ilLoadImage(argv[1]);

    int largeur, hauteur, bpp, format;

    largeur = ilGetInteger(IL_IMAGE_WIDTH);
    hauteur = ilGetInteger(IL_IMAGE_HEIGHT);
    bpp = ilGetInteger(IL_IMAGE_BYTES_PER_PIXEL);
    format = ilGetInteger(IL_IMAGE_FORMAT);

    unsigned char *donnees = ilGetData();
    unsigned char *nouvellesDonnees = new unsigned char[largeur * hauteur * bpp];

    int iterations = std::stoi(argv[2]);

    flouGaussienGPU(donnees, nouvellesDonnees, largeur, hauteur, bpp, iterations);

    ilTexImage(largeur, hauteur, 1, bpp, format, IL_UNSIGNED_BYTE, nouvellesDonnees);

    ilEnable(IL_FILE_OVERWRITE);
    ilSaveImage(argv[3]);

    ilDeleteImages(1, &image);
    delete[] nouvellesDonnees;

    return 0;
}

