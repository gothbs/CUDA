#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <IL/il.h>
#include <hip/hip_runtime.h>

// Le noyau du filtre gaussien
__constant__ float noyauGaussien[5][5] = {
    {1.0f, 4.0f, 7.0f, 4.0f, 1.0f},
    {4.0f, 16.0f, 26.0f, 16.0f, 4.0f},
    {7.0f, 26.0f, 41.0f, 26.0f, 7.0f},
    {4.0f, 16.0f, 26.0f, 16.0f, 4.0f},
    {1.0f, 4.0f, 7.0f, 4.0f, 1.0f}
};

// Appliquer le flou gaussien sur l'image en utilisant CUDA
__global__ void cudaFlouGaussien(unsigned char *donneesSrc, unsigned char *donneesDst, int largeur, int hauteur, int bpp) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    
    if (x >= 2 && y >= 2 && x + 2 < largeur && y + 2 < hauteur) {
        for (int c = 0; c < bpp; ++c) {
            float somme = 0.0f;
            for (int ky = -2; ky <= 2; ++ky) {
                for (int kx = -2; kx <= 2; ++kx) {
                    somme += donneesSrc[((y + ky) * largeur + (x + kx)) * bpp + c] * noyauGaussien[ky + 2][kx + 2];
                }
            }
            donneesDst[(y * largeur + x) * bpp + c] = static_cast<unsigned char>(somme);
        }
    }
}

void FlouGaussienGPU(unsigned char *donnees, unsigned char *nouvellesDonnees, int largeur, int hauteur, int bpp, int iterations, int blockSizeX, int blockSizeY) {
    unsigned char *donneesSrc = donnees;
    unsigned char *donneesDst = nouvellesDonnees;
    
    unsigned char *donneesSrcDevice;
    unsigned char *donneesDstDevice;
    
    size_t size = largeur * hauteur * bpp * sizeof(unsigned char);
    
    hipMalloc((void **)&donneesSrcDevice, size);
    hipMalloc((void **)&donneesDstDevice, size);
    
    hipMemcpy(donneesSrcDevice, donneesSrc, size, hipMemcpyHostToDevice);
    
    dim3 blockSize(blockSizeX, blockSizeY);
    dim3 gridSize((largeur + blockSize.x - 1) / blockSize.x, (hauteur + blockSize.y - 1) / blockSize.y);
    
    for (int i = 0; i < iterations; ++i) {
        cudaFlouGaussien<<<gridSize, blockSize>>>(donneesSrcDevice, donneesDstDevice, largeur, hauteur, bpp);
        std::swap(donneesSrcDevice, donneesDstDevice);
    }
    
    hipMemcpy(donneesDst, donneesSrcDevice, size, hipMemcpyDeviceToHost);
    
    hipFree(donneesSrcDevice);
    hipFree(donneesDstDevice);
}

int main(int argc, char *argv[]) {
    unsigned int image;

    // Initialisation de DevIL
    ilInit();
    ilGenImages(1, &image);
    ilBindImage(image);
    ilLoadImage(argv[1]);

    int largeur, hauteur, bpp, format;

    largeur = ilGetInteger(IL_IMAGE_WIDTH);
    hauteur = ilGetInteger(IL_IMAGE_HEIGHT);
    bpp = ilGetInteger(IL_IMAGE_BYTES_PER_PIXEL);
    format = ilGetInteger(IL_IMAGE_FORMAT);

    unsigned char *donnees = ilGetData();
    unsigned char *nouvellesDonnees = new unsigned char[largeur * hauteur * bpp];

    int iterations = std::stoi(argv[2]);

    // Normaliser le noyau du filtre gaussien
    float somme = 0.0f;
    for (int i = 0; i < 5; ++i) {
        for (int j = 0; j < 5; ++j) {
            somme += noyauGaussien[i][j];
        }
    }
    for (int i = 0; i < 5; ++i) {
        for (int j = 0; j < 5; ++j) {
            noyauGaussien[i][j] /= somme;
        }
    }

    // Appliquer le flou gaussien sur l'image avec le nombre d'itérations spécifié
    FlouGaussienGPU(donnees, nouvellesDonnees, largeur, hauteur, bpp, iterations, 16, 16);

    // Mettre à jour l'image avec les données traitées
    ilTexImage(largeur, hauteur, 1, bpp, format, IL_UNSIGNED_BYTE, nouvellesDonnees);

    // Activer l'écrasement de fichier lors de la sauvegarde
    ilEnable(IL_FILE_OVERWRITE);
    ilSaveImage(argv[3]);

    // Supprimer l'image de la mémoire de DevIL
    ilDeleteImages(1, &image);

    // Libérer la mémoire allouée pour les nouvelles données de l'image
    delete[] nouvellesDonnees;

    return 0;
}

